#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include "tensorUtil.h"
#include "sdt_alloc.h"

#define MAXDIM 8
#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))
#define MAX_THREADS_PER_BLOCK 1024
#define BLOCK_SIZE MAX_THREADS_PER_BLOCK

static float EPSILON = 1e-16;

/* static __device__ float E = 2.718281828; */

static int getIndex(int *ids, int ndim, int *dims)
{
     int i, id;
     for (i = 0, id = ids[0]; i < ndim-1; i++)
          id = dims[i+1] * id + ids[i+1];
     return id;
}

static void getIndexes(int id, int *ids, int ndim, int *dims)
{
     for (int i = ndim-1; i >=0; i--) {
          ids[i] = id % dims[i];
          id = id / dims[i];
     }
}

/* __global__ void sliceTensorKernel(uint8_t *src, uint8_t *dst, int sdim, int ddim, int start, int block_size) */
/* { */
/*      int di = blockIdx.x * block_size + threadIdx.x; */
/*      /\* si is the index of src elements to be copied. */
/*         The "block index" of src[si] is (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) *\/ */
/*      int si = (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) * block_size + threadIdx.x; */
/*      dst[di] = src[si]; */
/* } */

__global__ void sliceTensorKernel(uint8_t *src, uint8_t *dst, int start, int s_vol, int d_vol, int vol, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     int si = di / d_vol * s_vol + di % d_vol + start * vol;
     dst[di] = src[si];
}

__global__ void reduceArgMaxKernel(uint8_t *src, uint8_t *dst, uint8_t *arg, int dim_size, int reduce_vol, int batch_vol, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     /* src[si] is the first element in this thread to be compared, then
        si = batch_vol * batch + (di - reduce_vol * batch),
        where batch = di / reduce_vol,
        which is the same as the following code: */
     int si = (batch_vol - reduce_vol) * (di / reduce_vol) + di;
     uint8_t now = src[si], max = now;
     int maxi = 0;
     for (int i = 1; i < dim_size; i++) {
          now = src[si+i*reduce_vol];
          if (now > max) {
               max = now;
               maxi = i;
          }
     }
     dst[di] = max;
     arg[di] = maxi;
}

__global__ void multiplyElementKernel(uint8_t *src1, uint8_t *src2, uint8_t *dst, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     dst[di] = src1[di] * src2[di];
}

__global__ void transposeTensorKernel(uint8_t *src, uint8_t *dst, int ndim, int *s_dims, int *d_dims, int *s_ids, int *d_ids, int *axes, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     int *t_s_ids = s_ids + di * ndim;
     int *t_d_ids = d_ids + di * ndim;
     getIndexes(di, t_d_ids, ndim, d_dims);
     for (int i = 0; i < ndim; i++)
          t_s_ids[axes[i]] = t_d_ids[i];
     int si = getIndex(t_s_ids, ndim, s_dims);

     dst[di] = src[si];
}

__global__ void transformBboxSQDKernel(uint8_t *delta, uint8_t *anchor, uint8_t *res, float width, float height, float img_width, float img_height, int x_shift, int y_shift, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     /* int batch_idx = di / anchor_num; */
     /* now only support batch_size = 1 */
     float x_scale = 1.0 * img_width / width;
     float y_scale = 1.0 * img_height / height;

     /* (not used) si is the index of the first elements to be computed in the thread, then
        si = 4 * anchor_num * batch_idx + (di - anchor_num * batch_idx),
        which is the same as the following code: */
     /* int si = 3 * anchor_num * batch_idx  + di; */
     /* take 4 elements from each of delta and anchor */
     int si = di * 4;
     uint8_t d[4] = {delta[si], delta[si+1], delta[si+2], delta[si+3]};
     uint8_t a[4] = {anchor[si], anchor[si+1], anchor[si+2], anchor[si+3]};
     /* compute and put 4 result elements to res, according to SqueezeDet's source code */

     /* TODO: don't know why (maybe the resize), always has some shift compared to groundtruth*/
     uint8_t cx = (a[0] + d[0] * a[2]) * x_scale + x_shift;
     uint8_t cy = (a[1] + d[1] * a[3]) * y_scale + y_shift;
     uint8_t w = (a[2] * (d[2] < 1 ? expf(d[2]) : d[2] * E)) * x_scale;
     uint8_t h = (a[3] * (d[3] < 1 ? expf(d[3]) : d[3] * E)) * y_scale;
     res[si] = min(max(cx - w * 0.5, 0), img_width - 1);
     res[si+1] = min(max(cy - h * 0.5, 0), img_height - 1);
     res[si+2] = max(min(cx + w * 0.5, img_width - 1), 0);
     res[si+3] = max(min(cy + h * 0.5, img_height - 1), 0);
}

__global__ void pickElementsKernel(uint8_t *src, uint8_t *dst, int *idx, int stride, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     int si = idx[di];
     for (int i = 0; i < stride; i++)
          dst[di*stride+i] = src[si*stride+i];
}

static void assertTensor(const Tensor *tensor)
{
     assert(tensor && tensor->data);
     assert(tensor->ndim < MAXDIM && tensor->ndim > 0);
     assert(tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isTensorValid(const Tensor *tensor)
{
     return (tensor && tensor->data &&
             tensor->ndim < MAXDIM && tensor->ndim > 0 &&
             tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isShapeEqual(const Tensor *t1, const Tensor *t2)
{
     assertTensor(t1);
     assertTensor(t2);
     if (t1->ndim == t2->ndim) {
          int ndim = t1->ndim;
          while (--ndim >= 0)
               if (t1->dims[ndim] != t2->dims[ndim])
                    return 0;
          return 1;
     }
     return 0;
}

/* can only identify host memory alloced by hipHostMalloc, etc */
/* int isHostMem(const void *ptr) */
/* { */
/*      hipPointerAttribute_t attributes; */
/*      checkError(hipPointerGetAttributes(&attributes, ptr)); */
/*      return attributes.memoryType == hipMemoryTypeHost; */
/* } */

/* int isDeviceMem(const void *ptr) */
/* { */
/*      hipPointerAttribute_t attributes; */
/*      checkError(hipPointerGetAttributes(&attributes, ptr)); */
/*      return attributes.memoryType == hipMemoryTypeDevice; */
/* } */

void *cloneMem(const void *src, size_t size)
{
     assert(src);
     void *p;
     p = sdt_alloc(size);
     memmove(p, src, size);
     return p;
}

Tensor *cloneTensor(const Tensor *src)
{
     assert(isTensorValid(src));
     uint8_t *data = (uint8_t *)cloneMem(src->data, src->len * sizeof(uint8_t));
     Tensor *dst = createTensor(data, src->ndim, src->dims);
     return dst;
}

void *repeatMem(void *data, size_t size, int times)
{
     assert(data && times > 0);
     void *p, *dst;
     int i;
     dst = p = sdt_alloc(size * times);
     for (i = 0; i < times; i++, p = (char *)p + size * times)
          memmove(p, data, size);
     return dst;
}


int computeLength(int ndim, const int *dims)
{
     if (dims) {
          int i, len = 1;
          for (i = 0; i < ndim; i++)
               len *= dims[i];
          return len;
     }
     fprintf(stderr, "Warning: null dims in computeLength\n");
     return 0;
}

Tensor *createTensor(uint8_t *data, int ndim, const int *dims)
{
     Tensor *t = (Tensor *)sdt_alloc(sizeof(Tensor));
     t->data = data;
     t->ndim = ndim;
     t->dims = (int *)sdt_alloc(sizeof(int) * ndim);
     memmove(t->dims, dims, sizeof(int) * ndim);
     t->len = computeLength(ndim, dims);
     return t;
}

Tensor *mallocTensor(int ndim, const int* dims)
{
     Tensor *t = createTensor(NULL, ndim, dims);
     uint8_t *f;

     f = (uint8_t *)sdt_alloc(t->len * sizeof(uint8_t));
     t->data = f;

     return t;
}

void freeTensor(Tensor *t, int do_free_data)
{
     assert(isTensorValid(t));
     sdt_free(t->dims);
     if (do_free_data) {
          if (isDeviceMem(t->data))
               checkError(hipFree(t->data));
          else
               sdt_free(t->data);
     }
     sdt_free(t);
}

void fprintTensor(FILE *stream, const Tensor *tensor, const char *fmt)
{
     assertTensor(tensor);
     int dim_sizes[MAXDIM], dim_levels[MAXDIM]; /* dimision size and how deep current chars go */
     int ndim = tensor->ndim, len = tensor->len, *dims = tensor->dims; /* pointer short cut */
     uint8_t *data = tensor->data;
     char left_buf[MAXDIM+1], right_buf[MAXDIM+1]; /* buffer for brackets */
     char *lp = left_buf, *rp = right_buf;
     size_t right_len;
     int i, j, k;

     dim_sizes[ndim-1] = tensor->dims[ndim-1];
     dim_levels[ndim-1] = 0;
     for (i = ndim-2; i >= 0; i--) {
          dim_sizes[i] = dims[i] * dim_sizes[i+1];
          dim_levels[i] = 0;
     }
     for (i = 0; i < len; i++) {
          for (j = 0; j < ndim; j++) {
               if (i % dim_sizes[j] == 0)
                    dim_levels[j]++;
               if (dim_levels[j] == 1) {
                    *lp++ = '[';
                    dim_levels[j]++;
               }
               if (dim_levels[j] == 3) {
                    *rp++ = ']';
                    if (j != 0 && dim_levels[j] > dim_levels[j-1]) {
                         *lp++ = '[';
                         dim_levels[j] = 2;
                    } else
                         dim_levels[j] = 0;
               }
          }
          *lp = *rp = '\0';
          fprintf(stream, "%s", right_buf);
          if (*right_buf != '\0') {
               fprintf(stream, "\n");
               right_len = strlen(right_buf);
               for (k = ndim-right_len; k > 0; k--)
                    fprintf(stream, " ");
          }
          fprintf(stream, "%s", left_buf);
          if (*left_buf == '\0')
               fprintf(stream, " ");
          fprintf(stream, fmt, data[i]);
          lp = left_buf, rp = right_buf;
     }
     for (j = 0; j < ndim; j++)
          fprintf(stream, "]");
     fprintf(stream, "\n");
}

void printTensor(const Tensor *tensor, const char *fmt)
{
     fprintTensor(stdout, tensor, fmt);
}

/* void fprintDeviceTensor(FILE *stream, const Tensor *d_tensor, const char *fmt) */
/* { */
/*      assert(isTensorValid(d_tensor)); */
/*      Tensor *h_tensor = cloneTensor(d_tensor, D2H); */
/*      fprintTensor(stream, h_tensor, fmt); */
/*      free(h_tensor->data); /\* TODO: free t_tensor *\/ */
/* } */

/* void printDeviceTensor(const Tensor *d_tensor, const char *fmt) */
/* { */
/*      fprintDeviceTensor(stdout, d_tensor, fmt); */
/* } */

void saveTensor(const char *file_name, const Tensor *tensor, const char *fmt)
{
     FILE *fp = fopen(file_name, "w");
     fprintTensor(fp, tensor, fmt);
     fclose(fp);
}

/* void saveDeviceTensor(const char *file_name, const Tensor *d_tensor, const char *fmt) */
/* { */
/*      FILE *fp = fopen(file_name, "w"); */
/*      fprintDeviceTensor(fp, d_tensor, fmt); */
/*      fclose(fp); */
/* } */

/* Tensor *createSlicedTensor(const Tensor *src, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src)); */
/*      assert(dim <= src->ndim && dim >= 0); */
/*      assert(len+start <= src->dims[dim]); */

/*      Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor)); /\* new tensor *\/ */
/*      dst->ndim = src->ndim; */
/*      dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim); */
/*      memmove(dst->dims, src->dims, sizeof(int) * dst->ndim); */
/*      dst->dims[dim] = len; */
/*      dst->len = src->len / src->dims[dim] * len; */
/*      dst->data = (uint8_t *)sdt_alloc(dst->len * sizeof(uint8_t)); */
/*      return dst; */
/* } */

/* Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src) && isTensorValid(dst)); */
/*      assert(dst->ndim == src->ndim); */
/*      for (int i = 0; i < dst->ndim; i++) */
/*           assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]); */

/*      int i, block_size, block_num; /\* block size and number for copy operation *\/ */
/*      for (i = dim+1, block_size = 1; i < dst->ndim; i++) */
/*           block_size *= dst->dims[i]; */
/*      for (i = 0, block_num = 1; i <= dim; i++) */
/*           block_num *= dst->dims[i]; */

/*      int index; */
/*      uint8_t *dp = dst->data, *sp = src->data; */
/*      size_t uint8_ts_size = block_size * sizeof(uint8_t); */
/*      for (i = 0; i < block_num; i++) { */
/*           index = i / len * src->dims[dim] + i % len + start; */
/*           memmove(dp+i*block_size, sp+index*block_size, uint8_ts_size); */
/*      } */

/*      return dst; */
/* } */

Tensor *createSlicedTensor(const Tensor *src, int dim, int start, int len)
{
     assert(isTensorValid(src));
     assert(dim <= MAXDIM);
     assert(len+start <= src->dims[dim]);

     Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor)); /* new tensor */
     dst->ndim = src->ndim;
     dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = len;
     dst->len = src->len / src->dims[dim] * len;
     checkError(hipMalloc(&dst->data, sizeof(uint8_t) * dst->len));
     return dst;
}

/* Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src) && isTensorValid(dst)); */
/*      assert(isDeviceMem(src->data) && isDeviceMem(dst->data)); */
/*      assert(dst->ndim == src->ndim); */
/*      for (int i = 0; i < dst->ndim; i++) */
/*           assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]); */

/*      int i, block_size, block_num; /\* block size and number of cuda threads *\/ */
/*      int ddim = dst->dims[dim], sdim = src->dims[dim]; */
/*      for (i = dim+1, block_size = 1; i < dst->ndim; i++) */
/*           block_size *= dst->dims[i]; */
/*      for (i = 0, block_num = 1; i <= dim; i++) */
/*           block_num *= dst->dims[i]; */

/*      sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, sdim, ddim, start, block_size); */
/*      return dst; */
/* } */

Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len)
{
     assert(isTensorValid(src) && isTensorValid(dst));
     /* assert(isDeviceMem(src->data) && isDeviceMem(dst->data)); */
     assert(dst->ndim == src->ndim);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]);

     int i, d_vol, s_vol, vol;
     int thread_num, block_size, block_num; /* block size and number of cuda threads */
     for (i = dim+1, vol = 1; i < dst->ndim; i++)
          vol *= dst->dims[i];
     d_vol = vol * dst->dims[dim];
     s_vol = vol * src->dims[dim];
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     /* sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, start, s_vol, d_vol, vol, block_size, thread_num); */

     int si, di;
     for (di = 0; di < thread_num; di++) {
          si = di / d_vol * s_vol + di % d_vol + start * vol;
          dst[di] = src[si];
     }

     return dst;
}

/* in-place reshape tensor */
Tensor *reshapeTensor(const Tensor *src, int newNdim, const int *newDims)
{
     assert(isTensorValid(src));
     assert(newDims);
     assert(src->len == computeLength(newNdim, newDims));
     Tensor *dst = createTensor(src->data, newNdim, newDims); /* new tensor */
     return dst;
}

Tensor *createReducedTensor(const Tensor *src, int dim)
{
     assert(isTensorValid(src));
     assert(dim < src->ndim && dim >= 0);

     Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor));
     dst->ndim = src->ndim;
     dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = 1;
     dst->len = computeLength(dst->ndim, dst->dims);
     /* checkError(hipMalloc(&dst->data, sizeof(uint8_t) * dst->len)); */
     dst->data = (uint8_t *)sdt_alloc(sizeof(uint8_t) * dst->len);
     return dst;
}

void *reduceArgMax(const Tensor *src, Tensor *dst, Tensor *arg, int dim)
{
     assert(isTensorValid(src) && isTensorValid(dst) && isTensorValid(arg));
     /* assert(isDeviceMem(src->data) && isDeviceMem(dst->data) && isDeviceMem(arg->data)); */
     assert(dim < src->ndim && dim >= 0);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == 1 : dst->dims[i] == src->dims[i] &&
                 i == dim ? arg->dims[i] == 1 : arg->dims[i] == src->dims[i]);

     /* suppose the shape of src is [N, C, H, W], dim = 1, then thread_num is N x H x W
        reduce_vol is H x W, index_vol is C x H x W */
     int i, thread_num, block_size, block_num, reduce_vol, index_vol;
     for (i = dim+1, thread_num = 1; i < dst->ndim; i++)
          thread_num *= dst->dims[i];
     reduce_vol = thread_num;
     index_vol = thread_num * src->dims[dim];
     for (i = 0; i < dim; i++)
          thread_num *= dst->dims[i];
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     /* reduceArgMaxKernel<<<block_num, block_size>>>(src->data, dst->data, arg->data, src->dims[dim], reduce_vol, index_vol, block_size, thread_num); */

     int di, si;
     for (di = 0; di < thread_num; di++) {
          /* src[si] is the first element in this thread to be compared, then
             si = batch_vol * batch + (di - reduce_vol * batch),
             where batch = di / reduce_vol,
             which is the same as the following code: */
          si = (batch_vol - reduce_vol) * (di / reduce_vol) + di;
          uint8_t now = src[si], max = now;
          int maxi = 0;
          for (i = 1; i < dim_size; i++) {
               now = src[si+i*reduce_vol];
               if (now > max) {
                    max = now;
                    maxi = i;
               }
          }
          dst[di] = max;
          arg[di] = maxi;
     }

     return dst;
}

Tensor *multiplyElement(const Tensor *src1, const Tensor *src2, Tensor *dst)
{
     assert(isShapeEqual(src1, src2));
     assert(isShapeEqual(src1, dst));
     assert(isDeviceMem(src1->data) && isDeviceMem(src2->data) && isDeviceMem(dst->data));

     int thread_num, block_size, block_num;
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     /* multiplyElementKernel<<<block_num, block_size>>>(src1->data, src2->data, dst->data, block_size, dst->len); */

     int di, si;
     for (di = 0; di < thread_num; di++) {
          dst[di] = src1[di] * src2[di];
     }
     return dst;
}

/* (optional) workspace size equals (sizeof(int) * dst->ndim * dst->len), two of them */
Tensor *transposeTensor(const Tensor *src, Tensor *dst, int *axes, int **workspace)
{
     assert(isTensorValid(src) && isTensorValid(dst));
     assert(src->len == dst->len);
     assert(src->ndim == dst->ndim);

     int *s_ids, *d_ids, *s_dims, *d_dims;
     int thread_num, block_size, block_num;
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;
     s_dims = (int *)cloneMem(src->dims, sizeof(int) * src->ndim);
     d_dims = (int *)cloneMem(dst->dims, sizeof(int) * dst->ndim);
     if (!workspace) {
          s_ids = (int *)sdt_alloc(sizeof(int) * dst->ndim * thread_num);
          d_ids = (int *)sdt_alloc(sizeof(int) * dst->ndim * thread_num);
          /* checkError(hipMalloc(&s_ids, sizeof(int) * dst->ndim * thread_num)); */
          /* checkError(hipMalloc(&d_ids, sizeof(int) * dst->ndim * thread_num)); */
     } else {
          s_ids = workspace[0];
          d_ids = workspace[1];
     }

     /* transposeTensorKernel<<<block_num, block_size>>>(src->data, dst->data, dst->ndim, s_dims, d_dims, s_ids, d_ids, axes, block_size, thread_num); */

     int di, si;
     for (di = 0; di < thread_num; di++) {
          int *t_s_ids = s_ids + di * ndim;
          int *t_d_ids = d_ids + di * ndim;
          getIndexes(di, t_d_ids, ndim, d_dims);
          for (i = 0; i < ndim; i++)
               t_s_ids[axes[i]] = t_d_ids[i];
          int si = getIndex(t_s_ids, ndim, s_dims);

          dst[di] = src[si];
     }

     if (!workspace) {
          sdt_free(s_ids);
          sdt_free(d_ids);
          /* checkError(hipFree(s_ids)); */
          /* checkError(hipFree(d_ids)); */
     }
     sdt_free(s_dims);
     sdt_free(d_dims);
     /* checkError(hipFree(s_dims)); */
     /* checkError(hipFree(d_dims)); */
     return dst;
}

/* TODO: multiple type tensor */
/* transform from bbox delta to bbox coordinates, using hyper param EXP_THRESH = 1.0.
   delta, anchor, res are all of the same shape [..., 4]
   width and height are resized image width and height.
   x_scales and y_scales are (temporary) pointers to width/original_width and height/original_height. */
Tensor *transformBboxSQD(const Tensor *delta, const Tensor *anchor, Tensor *res, int width, int height, int img_width, int img_height)
{
     assert(isShapeEqual(delta, anchor));
     assert(isShapeEqual(delta, res));
     assert(delta->ndim == 5);
     assert(delta->dims[4] == 4);
     /* assert(isDeviceMem(delta->data) && isDeviceMem(anchor->data) && isDeviceMem(res->data)); */

     /* take 4 elements from each of delta and anchor,
        and put 4 result elements to res in one thread */
     int i, thread_num, block_size, block_num;
     for (i = 0, thread_num = 1; i < res->ndim-1; i++)
          thread_num *= res->dims[i];
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     /* transformBboxSQDKernel<<<block_num, block_size>>>(delta->data, anchor->data, res->data, width, height, img_width, img_height, x_shift, y_shift, block_size, thread_num); */

     int di, si;
     for (di = 0; di < thread_num; di++) {
          /* int batch_idx = di / anchor_num; */
          /* now only support batch_size = 1 */
          float x_scale = 1.0 * img_width / width;
          float y_scale = 1.0 * img_height / height;

          /* (not used) si is the index of the first elements to be computed in the thread, then
             si = 4 * anchor_num * batch_idx + (di - anchor_num * batch_idx),
             which is the same as the following code: */
          /* int si = 3 * anchor_num * batch_idx  + di; */
          /* take 4 elements from each of delta and anchor */
          int si = di * 4;
          uint8_t d[4] = {delta[si], delta[si+1], delta[si+2], delta[si+3]};
          uint8_t a[4] = {anchor[si], anchor[si+1], anchor[si+2], anchor[si+3]};
          /* compute and put 4 result elements to res, according to SqueezeDet's source code */

          /* TODO: don't know why (maybe the resize), always has some shift compared to groundtruth*/
          uint8_t cx = (a[0] + d[0] * a[2]) * x_scale + x_shift;
          uint8_t cy = (a[1] + d[1] * a[3]) * y_scale + y_shift;
          uint8_t w = (a[2] * (d[2] < 1 ? expf(d[2]) : d[2] * E)) * x_scale;
          uint8_t h = (a[3] * (d[3] < 1 ? expf(d[3]) : d[3] * E)) * y_scale;
          res[si] = min(max(cx - w * 0.5, 0), img_width - 1);
          res[si+1] = min(max(cy - h * 0.5, 0), img_height - 1);
          res[si+2] = max(min(cx + w * 0.5, img_width - 1), 0);
          res[si+3] = max(min(cy + h * 0.5, img_height - 1), 0);
     }
     return res;
}

void tensorIndexSort(Tensor *src, int *idx)
{
     assert(isTensorValid(src));
     assert(idx);
     assert(isDeviceMem(src->data) && isDeviceMem(idx));

     /* the thrust call below can be unreliable, sometimes produces error */
     /* now it works with compilation flag -arch=sm_35 */
     /* TODO: replace thrust call by our own kernel */
     /* thrust::sort_by_key(thrust::device, src->data, src->data + src->len, idx, thrust::greater<uint8_t>()); */
}

void pickElements(uint8_t *src, uint8_t *dst, int stride, int *idx, int len)
{
     assert(src && dst && idx);
     assert(isDeviceMem(src) && isDeviceMem(dst) && isDeviceMem(idx));

     int thread_num, block_size, block_num;
     thread_num = len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     pickElementsKernel<<<block_num, block_size>>>(src, dst, idx, stride, block_size, thread_num);
}

/* void pickElements(uint8_t* src,uint8_t* dst,int stride,int* idx,int len) */
/* { */
/*      assert(src && dst && idx); */

/*      for (int i = 0; i < len; i++) { */
/*           for (int j = 0; j < stride; j++) { */
/*                fprintf(stderr, "i: %d j: %d idx[i]: %d src[idx[i]]: %.2f", */
/*                        i, j, idx[i], src[idx[i]]); */
/*                fprintf(stderr, "\n"); */
/*                dst[i*stride+j] = src[idx[i]*stride+j]; */
/*           } */
/*      } */
/* } */

/* compute the iou of two bboxes whose elements are {top_left_x, top_left_y, bottom_right_x, bottom_right_y} */
float computeIou(float *bbox0, float *bbox1)
{
     assert(bbox0 && bbox1);

     float lr, tb;              /* left-right, top-bottom for intersection*/
     float intersection, total;
     lr = min(bbox0[2], bbox1[2]) - max(bbox0[0], bbox1[0]);
     if (lr >= 0) {
          tb = min(bbox0[3], bbox1[3]) - max(bbox0[1], bbox1[1]);
          if (tb >= 0) {
               intersection = tb * lr + EPSILON;
               total = (bbox0[2] - bbox0[0]) * (bbox0[3] - bbox0[1]) +
                    (bbox1[2] - bbox1[0]) * (bbox1[3] - bbox1[1]) - intersection;
               return intersection / (total + EPSILON);
          }
     }
     return 0;
}
